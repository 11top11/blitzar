#include "hip/hip_runtime.h"
#include "benchmark/multi_exp1/multi_exp_gpu.h"

#include "benchmark/multi_exp1/multiply_add.h"
#include "sxt/curve21/constant/zero.h"
#include "sxt/curve21/operation/add.h"
#include "sxt/curve21/type/element_p3.h"

namespace sxt {
constexpr int num_threads_v = 128;

//--------------------------------------------------------------------------------------------------
// compute_reduction 
//--------------------------------------------------------------------------------------------------
__device__ static void compute_reduction(c21t::element_p3& res_mi,
                                         c21t::element_p3* reductions, int mi,
                                         int first, int last) {
  auto tid = threadIdx.x;
  auto& reduction = reductions[tid];
  reduction = c21cn::zero_p3_v;
  for (int i=first; i<last; i+=num_threads_v) {
    multiply_add(reduction, mi, i);
  }

  __syncthreads();
  for (int s=num_threads_v/2; s>0; s>>=1) {
    if (tid < s) {
      c21o::add(reduction, reduction, reductions[tid + s]);
    }
    __syncthreads();
  }
  if (tid == 0) {
    res_mi = reduction;
  }
}


//--------------------------------------------------------------------------------------------------
// multi_exp_kernel 
//--------------------------------------------------------------------------------------------------
__global__ static void multi_exp_kernel(c21t::element_p3* res, int n) {
  __shared__ c21t::element_p3 reductions[num_threads_v];
  auto first = threadIdx.x;
  int mi = blockIdx.x;
  compute_reduction(res[mi], reductions, mi, first, n);
}

//--------------------------------------------------------------------------------------------------
// multi_exp_gpu
//--------------------------------------------------------------------------------------------------
void multi_exp_gpu(c21t::element_p3* res, int m, int n) noexcept {
  c21t::element_p3* device_elements;
  hipMalloc(&device_elements, m * sizeof(c21t::element_p3));

  multi_exp_kernel<<<m, num_threads_v>>>(device_elements, n);

  hipMemcpy(res, device_elements, m * sizeof(c21t::element_p3),
             hipMemcpyDeviceToHost);
  hipFree(device_elements);
}
} // namespace sxt

